#include "hip/hip_runtime.h"
#include "matrix.cuh"

__device__ matrix_t* device_matrix_constructor(buffer_t* buffer, unsigned int rows, unsigned int cols)
{
	//assert(rows > 0 && cols > 0);
	
	matrix_t* m = (matrix_t*)buffer_malloc(buffer, sizeof(matrix_t) + sizeof(float) * rows * cols);

	m->rows = rows;
	m->cols = cols;
	device_set_matrix(m, 0.0);

	return m;
}

__device__ matrix_t* device_matrix_add(buffer_t* buffer, matrix_t* m1, matrix_t* m2)
{
	//assert(m1 != NULL && m2 != NULL);
	//assert(m1->rows > 0 && m2->rows > 0 && m1->cols > 0 && m2->cols > 0);
	//assert(m1->rows == m2->rows && m1->cols == m2->cols);

	matrix_t* sum = device_matrix_constructor(buffer, m1->rows, m1->cols);

	int i, j;
	for(i=0; i<m1->rows; i++)
	{
		for(j=0; j<m1->cols; j++)
		{
			device_matrix_set(sum, i, j, device_matrix_get(m1, i, j) + device_matrix_get(m2, i, j));
		}
	}
	return sum;
}

__device__ matrix_t* device_matrix_subtract(buffer_t* buffer, matrix_t* m1, matrix_t* m2)
{
	//assert(m1 != NULL && m2 != NULL);
	//assert(m1->rows > 0 && m2->rows > 0 && m1->cols > 0 && m2->cols > 0);
	//assert(m1->rows == m2->rows && m1->cols == m2->cols);

	matrix_t* difference = device_matrix_constructor(buffer, m1->rows, m1->cols);

	int i, j;
	for(i=0; i<m1->rows; i++)
	{
		for(j=0; j<m1->cols; j++)
		{
			device_matrix_set(difference, i, j, device_matrix_get(m1, i, j) - device_matrix_get(m2, i , j));
		}
	}
	return difference;
}

__device__ matrix_t* device_matrix_multiply(buffer_t* buffer, matrix_t* m1, matrix_t* m2)
{
	if(!(m1->rows > 0 && m2->rows > 0 && m1->cols > 0 && m2->cols > 0))
	{
		//printf("%d %d %d %d", m1->rows, m2->rows, m1->cols, m2->cols);
	}
	//assert(m1 != NULL && m2 != NULL);
	//assert(m1->rows > 0 && m2->rows > 0 && m1->cols > 0 && m2->cols > 0);
	//assert(m1->cols == m2->rows);

	matrix_t* product = device_matrix_constructor(buffer, m1->rows, m2->cols);

	int i, j, k;
	for(i=0; i<product->rows; i++)
	{
		for(j=0; j<product->cols; j++)
		{
			for(k=0; k<m1->cols; k++)
			{
				device_matrix_set(product, i, j, device_matrix_get(product, i, j) + device_matrix_get(m1, i, k) * device_matrix_get(m2, k, j));
			}
		}
	}
	return product;
}

__device__ matrix_t* device_matrix_scalar_multiply(buffer_t* buffer, matrix_t* m, float scalar)
{
	//assert(m!= NULL);
	//assert(m->rows > 0 && m->cols > 0);

	matrix_t* product = device_matrix_constructor(buffer, m->rows, m->cols);

	int i, j;
	for(i=0; i<m->rows; i++)
	{
		for(j=0; j<m->cols; j++)
		{
			device_matrix_set(product, i, j, device_matrix_get(m, i, j) * scalar);
		}
	}
	return product;
}

__device__ matrix_t* device_matrix_sigmoid(buffer_t* buffer, matrix_t* m)
{
	matrix_t* copy = device_copy_matrix(buffer, m);
	int i, j;
	for(i=0; i<m->rows; i++)
	{
		for(j=0; j<m->cols; j++)
		{
			device_matrix_set(copy, i, j, 1.0 / (1.0 + exp(-1.0 * device_matrix_get(copy, i, j))));
		}
	}
	return copy;
}

__device__ matrix_t* device_matrix_sigmoid_gradient(buffer_t* buffer, matrix_t* m)
{
	float sig;
	matrix_t* copy = device_copy_matrix(buffer, m);
	int i, j;
	for(i=0; i<m->rows; i++)
	{
		for(j=0; j<m->cols; j++)
		{
			sig = 1.0 / (1.0 + exp(-1.0 * device_matrix_get(copy, i, j)));
			device_matrix_set(copy, i, j, sig * (1-sig));
		}
	}
	return copy;
}

__device__ matrix_t* device_matrix_square(buffer_t* buffer, matrix_t* m)
{
	matrix_t* copy = device_copy_matrix(buffer, m);
	int i, j;
	for(i=0; i<m->rows; i++)
	{
		for(j=0; j<m->cols; j++)
		{
			device_matrix_set(copy, i, j, pow(device_matrix_get(copy, i, j), 2));
		}
	}
	return copy;
}

__device__ matrix_t* device_matrix_cell_multiply(buffer_t* buffer, matrix_t* m1, matrix_t* m2)
{
	//assert(m1 != NULL && m2 != NULL);
	//assert(m1->rows > 0 && m2->rows > 0 && m1->cols > 0 && m2->cols > 0);
	//assert(m1->rows == m2->rows && m1->cols == m2->cols);

	matrix_t* product = device_matrix_constructor(buffer, m1->rows, m1->cols);

	int i, j;
	for(i=0; i<m1->rows; i++)
	{
		for(j=0; j<m1->cols; j++)
		{
			device_matrix_set(product, i, j, device_matrix_get(m1, i, j) * device_matrix_get(m2, i , j));
		}
	}
	return product;
}

__device__ matrix_t* device_matrix_transpose(buffer_t* buffer, matrix_t* m)
{
	//assert(m!= NULL);
	//assert(m->rows > 0 && m->cols > 0);

	matrix_t* transpose = device_copy_matrix(buffer, m);
	transpose->rows = m->cols;
	transpose->cols = m->rows;
	int i, j;
	for(i=0; i<m->rows; i++)
	{
		for(j=0; j<m->cols; j++)
		{
			device_matrix_set(transpose, j, i, device_matrix_get(m, i, j));
		}
	}
	return transpose;
}

__device__ matrix_t* device_copy_matrix(buffer_t* buffer, matrix_t* m)
{
	matrix_t* copy = device_matrix_constructor(buffer, m->rows, m->cols);
	memcpy(copy->matrix, m->matrix, sizeof(float)*m->rows*m->cols);
	return copy;
}

__device__ void device_free_matrix(matrix_t* m)
{
}

__device__ float device_matrix_get(matrix_t* m, unsigned int x, unsigned int y)
{
	//assert(m != NULL);
	////assert(x >= 0 && x < m->rows && y >= 0 && y < m->cols);
	return (m->matrix[x * m->cols + y]);
}

__device__ void device_matrix_set(matrix_t* m, unsigned int x, unsigned int y, float value)
{
	//assert(m != NULL);
	////assert(x >= 0 && x < m->rows && y >= 0 && y < m->cols);
	m->matrix[x * m->cols + y] = value;
}

__device__ void device_set_matrix(matrix_t* m, float val)
{
	//assert(m != NULL);
	//assert(m->rows > 0 && m->cols > 0);

	int i, j;
	for(i=0; i<m->rows; i++)
	{
		for(j=0; j<m->cols; j++)
		{
			device_matrix_set(m, i, j, val);
		}
	}
}

__device__ void device_set_matrix_index(matrix_t* m)
{
	//assert(m != NULL);
	//assert(m->rows > 0 && m->cols > 0);

	int i, j;
	for(i=0; i<m->rows; i++)
	{
		for(j=0; j<m->cols; j++)
		{
			device_matrix_set(m, i, j, i * m->cols + j);
		}
	}
}

__device__ unsigned int device_matrix_memory_size(matrix_t* m)
{
	return sizeof(matrix_t) + sizeof(float) * m->rows * m->cols;
}

__device__ unsigned int device_matrix_list_memory_size(matrix_list_t* m)
{
	unsigned int memory_size = sizeof(matrix_list_t);
	unsigned int i;
	for(i=0; i<m->num; i++)
	{
		memory_size += device_matrix_memory_size(m->matrix_list[i]);
	}
	return memory_size;
}

__device__ matrix_t* device_row_to_vector(buffer_t* buffer, matrix_t* m, unsigned int row)
{
	matrix_t* v = device_matrix_constructor(buffer, 1, m->cols);
	unsigned int i;
	for(i=0; i<m->cols; i++)
	{
		device_vector_set(v, i, device_matrix_get(m, row, i));
	}
	return v;
}

__device__ matrix_t* device_col_to_vector(buffer_t* buffer, matrix_t* m, unsigned int col)
{
	matrix_t* v = device_matrix_constructor(buffer, 1, m->rows);
	unsigned int i;
	for(i=0; i<m->rows; i++)
	{
		device_vector_set(v, i, device_matrix_get(m, i, col));
	}
	return v;
}

__device__ matrix_t* device_matrix_prepend_col(buffer_t* buffer, matrix_t* m, float value)
{
	matrix_t* result = device_matrix_constructor(buffer, m->rows, m->cols+1);
	unsigned int i, j;
	for(i=0; i<result->rows; i++)
	{
		device_matrix_set(result, i, 0, value);
	}
	for(i=0; i<m->rows; i++)
	{
		for(j=0; j<m->cols; j++)
		{
			device_matrix_set(result, i, j+1, device_matrix_get(m, i, j));
		}
	}
	return result;
}

__device__ matrix_t* device_matrix_remove_col(buffer_t* buffer, matrix_t* m)
{
	matrix_t* result = device_matrix_constructor(buffer, m->rows, m->cols-1);
	unsigned int i, j;
	for(i=0; i<result->rows; i++)
	{
		for(j=0; j<result->cols; j++)
		{
			device_matrix_set(result, i, j, device_matrix_get(m, i, j+1));
		}
	}
	return result;
}

__device__ matrix_t* device_matrix_prepend_row(buffer_t* buffer, matrix_t* m, float value)
{
	matrix_t* result = device_matrix_constructor(buffer, m->rows+1, m->cols);
	unsigned int i, j;
	for(i=0; i<result->cols; i++)
	{
		device_matrix_set(result, 0, i, value);
	}
	for(i=0; i<m->rows; i++)
	{
		for(j=0; j<m->cols; j++)
		{
			device_matrix_set(result, i+1, j, device_matrix_get(m, i, j));
		}
	}
	return result;
}

__device__ matrix_t* device_matrix_remove_row(buffer_t* buffer, matrix_t* m)
{
	matrix_t* result = device_matrix_constructor(buffer, m->rows-1, m->cols);
	unsigned int i, j;
	for(i=0; i<result->rows; i++)
	{
		for(j=0; j<result->cols; j++)
		{
			device_matrix_set(result, i, j, device_matrix_get(m, i+1, j));
		}
	}
	return result;
}

__device__ void device_matrix_add_to(matrix_t* m1, matrix_t* m2)
{
	int i, j;
	for(i=0; i<m1->rows; i++)
	{
		for(j=0; j<m1->cols; j++)
		{
			device_matrix_set(m1, i, j, device_matrix_get(m1, i, j) + device_matrix_get(m2, i, j));
		}
	}
}
