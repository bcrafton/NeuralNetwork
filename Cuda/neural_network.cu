#include "hip/hip_runtime.h"
#include "neural_network.cuh"

#define ALPHA .5

__device__ void calculate_gradient_at(buffer_t* buffer, int index, matrix_list_t** gradient, matrix_list_t* theta, unsigned int num_layers, unsigned int num_labels,
		matrix_t* X, matrix_t* y, double lamda)
{
	unsigned int i;

	matrix_list_t* local_gradient = device_matrix_list_constructor(buffer, theta->num);
	for(i=0; i<local_gradient->num; i++)
	{
		local_gradient->matrix_list[i] = device_matrix_constructor(buffer, theta->matrix_list[i]->rows, theta->matrix_list[i]->cols);
	}

	matrix_t* temp;
	matrix_t* temp2;
	matrix_t* temp3;

	matrix_list_t* A = device_matrix_list_constructor(buffer, num_layers);
	matrix_list_t* Z = device_matrix_list_constructor(buffer, num_layers-1);
	matrix_list_t* delta = device_matrix_list_constructor(buffer, num_layers-1);

	A->matrix_list[0] = device_row_to_vector(buffer, X, index);
	temp = device_matrix_prepend_col(buffer, A->matrix_list[0], 1.0);

	A->matrix_list[0] = device_matrix_transpose(buffer, temp);

	for(i=0; i<num_layers-1; i++)
	{
		Z->matrix_list[i] = device_matrix_multiply(buffer, theta->matrix_list[i], A->matrix_list[i]);

		temp = device_matrix_sigmoid(buffer, Z->matrix_list[i]);
		A->matrix_list[i+1] = device_matrix_prepend_row(buffer, temp, 1.0);
	}

	temp = device_matrix_remove_row(buffer, A->matrix_list[num_layers-1]);
	A->matrix_list[num_layers-1] = temp;
	
	matrix_t* result_matrix = device_matrix_constructor(buffer, 1, num_labels);
	for(i = 0; i < num_labels; i++)
	{
		if(device_vector_get(y, index) == i)
		{
			device_vector_set(result_matrix, i, 1.0);
		}
	}
	temp = device_matrix_transpose(buffer, result_matrix);
	result_matrix= temp;

	delta->matrix_list[1] = device_matrix_subtract(buffer, A->matrix_list[num_layers-1], result_matrix);
	
	matrix_t* theta_transpose = device_matrix_transpose(buffer, theta->matrix_list[1]);
	temp = device_matrix_multiply(buffer, theta_transpose, delta->matrix_list[1]);

	matrix_t* sig_gradient = device_matrix_sigmoid_gradient(buffer, Z->matrix_list[0]);
	temp2 = device_matrix_prepend_row(buffer, sig_gradient, 1.0);

	temp3 = device_matrix_cell_multiply(buffer, temp, temp2);
	delta->matrix_list[0] = device_matrix_remove_row(buffer, temp3);

	for(i=0; i<num_layers-1; i++)
	{
		matrix_t* A_transpose = device_matrix_transpose(buffer, A->matrix_list[i]);
		temp = device_matrix_multiply(buffer, delta->matrix_list[i], A_transpose);
		temp2 = device_matrix_add(buffer, local_gradient->matrix_list[i], temp);
		local_gradient->matrix_list[i] = temp2;
	}

	*gradient = local_gradient;
}

__global__ void calculate_gradient_kernel(void* gradient, void* memptr, size_t size, matrix_t* rolled_theta, unsigned int num_layers, unsigned int num_labels,
		matrix_t* X, matrix_t* y, double lamda)
{
	unsigned int layer_sizes[][2] = {{25, 401}, {10, 26}};

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < 5000)
	{
		buffer_t* buffer = buffer_constructor(size, memptr+(tid * size));	
		
		matrix_list_t* local_gradient;
		matrix_list_t* theta = device_unroll_matrix_list(buffer, rolled_theta, num_layers-1, layer_sizes);
		
		
		//calculate_gradient_at(buffer, tid, &local_gradient, theta, num_layers, num_labels, X, y, lamda);

		matrix_t* result = (matrix_t*) (gradient + (tid * device_matrix_memory_size(rolled_theta)));

		matrix_t* tmp = device_roll_matrix_list(buffer, theta);
		memcpy(result, tmp, device_matrix_memory_size(rolled_theta));
	}
}

void calculate_gradient(matrix_list_t** gradient, matrix_list_t* theta, unsigned int num_layers, unsigned int num_labels,
		matrix_t* X, matrix_t* y, double lamda)
{
	unsigned int layer_sizes[][2] = {{25, 401}, {10, 26}};

	unsigned int m = X->rows;
	//unsigned int n = X->cols;
	unsigned int i, j;

	void* memptr;
	void* device_gradient;
	matrix_t* device_rolled_theta;
	matrix_t* device_X;
	matrix_t* device_y;
	
	matrix_t* rolled_theta = roll_matrix_list(theta);

	hipMalloc(&memptr, 262144*5000);
	hipMalloc(&device_gradient, matrix_memory_size(rolled_theta)*5000);
	hipMalloc(&device_rolled_theta, matrix_memory_size(rolled_theta));
	hipMalloc(&device_X, matrix_memory_size(X));
	hipMalloc(&device_y, matrix_memory_size(y));
	
	hipMemcpy(device_X, X, matrix_memory_size(X), hipMemcpyHostToDevice);
	hipMemcpy(device_y, y, matrix_memory_size(y), hipMemcpyHostToDevice);
	hipMemcpy(device_rolled_theta, rolled_theta, matrix_memory_size(rolled_theta), hipMemcpyHostToDevice);
	
	int block_size = 1024;
	int grid_size = 5000 / block_size;
	if(5000 % block_size)
	{
		grid_size = grid_size + 1;
	}
	
	calculate_gradient_kernel<<<grid_size, block_size>>>(device_gradient, memptr, 262144, device_rolled_theta, num_layers, num_labels, device_X, device_y, lamda);
	
	matrix_t* rolled_gradient = matrix_constructor(rolled_theta->rows, rolled_theta->cols);
	hipMemcpy(rolled_gradient, device_gradient, matrix_memory_size(rolled_theta), hipMemcpyDeviceToHost);
	matrix_list_t* gradient_sum = unroll_matrix_list(rolled_gradient, num_layers-1, layer_sizes);	

	matrix_t* temp;
	matrix_t* temp2;
	matrix_t* temp3;
	
	for(i=0; i<num_layers-1; i++)
	{
		temp = matrix_scalar_multiply(gradient_sum->matrix_list[i], 1.0/m);
		temp2 = copy_matrix(theta->matrix_list[i]);
		for(j=0; j<theta->matrix_list[i]->rows; j++)
		{
			matrix_set(temp2, j, 0, 0.0);
		}
		free_matrix(gradient_sum->matrix_list[i]);
		temp3 = matrix_scalar_multiply(temp2, lamda/m);
		gradient_sum->matrix_list[i] = matrix_add(temp, temp3);
		free_matrix(temp);
		free_matrix(temp2);
		free_matrix(temp3);
	}

	*gradient = gradient_sum;

	hipFree(memptr);
	hipFree(device_gradient);
	hipFree(device_rolled_theta);
	hipFree(device_X);
	hipFree(device_y);
	
}


void gradient_descent(matrix_list_t** theta, unsigned int num_layers, unsigned int num_labels, matrix_t* X, matrix_t* y,
		double lamda, unsigned int iteration_number)
{
	clock_t start, end;
	double cpu_time_used;
	start = clock();

	matrix_list_t* gradient;

	unsigned int i;
	for(i=0; i < iteration_number; i++)
	{
		calculate_gradient(&gradient, *theta, num_layers, num_labels, X, y, lamda);
		
		matrix_list_t* tmp;
		tmp = matrix_list_scalar_multiply(gradient, ALPHA);
		free_matrix_list(gradient);
		gradient = tmp;

		tmp = matrix_list_subtract(*theta, gradient);
		free_matrix_list(*theta);
		*theta = tmp;

		free_matrix_list(gradient);

		if((i+1) % 10 == 0)
		{
			end = clock();
			cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
			printf("iteration #%d, accuracy: %f, time used: %f\n", i+1, accuracy(*theta, X, y), cpu_time_used);
		}
	}
}


matrix_list_t* random_init_weights(unsigned int num_layers, unsigned int layer_sizes[])
{
	srand(time(NULL));

	matrix_list_t* theta = matrix_list_constructor(num_layers-1);
	unsigned int i, j, k;
	for(i = 0; i<num_layers-1; i++)
	{
		theta->matrix_list[i] = matrix_constructor(layer_sizes[i+1], layer_sizes[i]+1);
		for(j=0; j<theta->matrix_list[i]->rows; j++)
		{
			for(k = 0; k<theta->matrix_list[i]->cols; k++)
			{
				double random_double = ((double)(rand() % 1000)) / (double)1000;
				matrix_set(theta->matrix_list[i], j, k, random_double * 2 * .12 - .12);
			}
		}
	}
	return theta;
}

double accuracy(matrix_list_t* theta, matrix_t* X, matrix_t* y)
{
	assert(theta->num == 2);
	matrix_t* theta_transpose, *temp, *temp2;

	theta_transpose = matrix_transpose(theta->matrix_list[0]);
	temp = matrix_prepend_col(X, 1.0);
	temp2 = matrix_multiply(temp, theta_transpose);
	matrix_t* h1 = matrix_sigmoid(temp2);

	free_matrix(theta_transpose);
	free_matrix(temp);
	free_matrix(temp2);

	theta_transpose = matrix_transpose(theta->matrix_list[1]);
	temp = matrix_prepend_col(h1, 1.0);
	temp2 = matrix_multiply(temp, theta_transpose);
	matrix_t* h2 = matrix_sigmoid(temp2);

	free_matrix(theta_transpose);
	free_matrix(temp);
	free_matrix(temp2);

	assert(h2->rows == 5000 && h2->cols == 10);
	matrix_t* p = matrix_constructor(1, 5000);
	int i, j;

	for(i = 0; i<h2->rows; i++)
	{
		double max = 0.0;
		unsigned char first = 1;
		for(j=0; j<h2->cols; j++)
		{
			if(matrix_get(h2, i, j) > max || first == 1)
			{
				vector_set(p, i, j);
				max = matrix_get(h2, i, j);
				first = 0;
			}
		}
	}
	double count = 0;
	for(i=0; i<5000; i++)
	{
		if(vector_get(y, i) == vector_get(p, i))
			count = count + 1;
	}

	free_matrix(p);
	free_matrix(h1);
	free_matrix(h2);
	
	return count/5000;
}
